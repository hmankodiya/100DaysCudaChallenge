#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<string>
#include<math.h>

#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include "../include/utils.cuh"

__global__ void vector_vector_add(float *d_x, float *d_y, float *d_result, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
        d_result[index] = d_x[index] + d_y[index];
}

void verify_result(const float *h_result, const float *h_x, const float *h_y, int N) {
    bool all_correct = true;
    for (int i = 0;i < N;i++) {
        float expected = h_x[i] + h_y[i];
        if (fabs(h_result[i] - expected) > TOLERANCE) {
            std::cerr << "❌ Mismatch at index " << i
                << ": got " << h_result[i]
                << ", expected " << expected << std::endl;
            all_correct = false;
            break;
        }
    }

    if (all_correct) {
        std::cout << "✅ Result verification passed!" << std::endl;
    }
    else {
        std::cout << "❌ Result verification failed." << std::endl;
    }
}

float run_vector_vector_add(float *d_x, float *d_y, float *d_result, int N) {
    dim3 blockDim(256);  // reasonable block size
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);  // ceil(N / blockDim.x)

    float ms = 0.0f;
    hipEvent_t start, stop;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));
    vector_vector_add << <gridDim, blockDim >> > (d_x, d_y, d_result, N);
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));

    printf("------- Vector Vector Add Kernel ---------\n");
    printf("Elapsed time: %f ms\n", ms);
    printf("------------------------------------------\n");

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return ms;
}