#include <iostream>
#include<stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../include/utils.cuh"
#include "./include/blockwise_sum.cuh"


int main() {
    // Matrix dimensions
    int N;
    printf("array size:");
    scanf("%d", &N);

    int block_dim;
    printf("block_dim:");
    scanf("%d", &block_dim);

    const size_t size = N * sizeof(float);

    // Allocate and initialize host matrices
    float *h_A = arange(0, N, N);
    if (N < 20)
        display(h_A, N);

    dim3 blockDim(block_dim);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);

    // Allocate device memory
    float *d_A, *d_result, *h_result;
    size_t size_result = gridDim.x * sizeof(float);

    CUDA_CHECK(hipMalloc((void **)&d_A, size));
    CUDA_CHECK(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice)); // Copy data from host to device

    h_result = (float *)malloc(size_result);
    CUDA_CHECK(hipMalloc((void **)&d_result, size_result));

    // Run the kernel
    float ms = run_blockwise_sum(d_A, d_result, N, block_dim);

    CUDA_CHECK(hipMemcpy(h_result, d_result, size_result, hipMemcpyDeviceToHost)); // Copy data from host to device

    // blockwise sum results
    if (N < 20)
        display(h_result, gridDim.x);

    verify_result(h_result, h_A, N, gridDim.x);

    // Cleanup
    free(h_A);
    free(h_result);
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_result));

    return 0;
}
