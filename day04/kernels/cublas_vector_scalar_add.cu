#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "../include/utils.cuh"

void verify_result(const float *h_result, const float *h_x, const float scalar, int N) {
    bool all_correct = true;
    for (int i = 0;i < N;i++) {
        float expected = h_x[i] + scalar;
        if (fabs(h_result[i] - expected) > TOLERANCE) {
            std::cerr << "❌ Mismatch at index " << i
                << ": got " << h_result[i]
                << ", expected " << expected << std::endl;
            all_correct = false;
            break;
        }
    }

    if (all_correct) {
        std::cout << "✅ Result verification passed!" << std::endl;
    }
    else {
        std::cout << "❌ Result verification failed." << std::endl;
    }
}


float *cublas_vector_scalar_add(hipblasHandle_t handle, float *d_x, float scalar, int N) {
    float *h_ones = ones_arr(N);  // host ones array
    size_t size = N * sizeof(float);

    float *d_ones;
    CUDA_CHECK(hipMalloc((void **)&d_ones, size));
    CUDA_CHECK(hipMemcpy(d_ones, h_ones, size, hipMemcpyHostToDevice));

    // Perform: d_x = d_x + scalar * d_ones
    CUBLAS_CHECK(hipblasSaxpy(handle, N, &scalar, d_ones, 1, d_x, 1));

    // Allocate space for result on host and copy back
    float *h_result = (float *)malloc(size);
    CUDA_CHECK(hipMemcpy(h_result, d_x, size, hipMemcpyDeviceToHost));

    // Cleanup
    free(h_ones);
    CUDA_CHECK(hipFree(d_ones));

    return h_result;  // Caller is responsible for freeing h_result
}


float run_vector_scalar_add(float *d_x, float scalar, int N) {
    // dim3 blockDim(256);
    // dim3 gridDim((N + blockDim.x - 1) / blockDim.x);

    float ms = 0.0f;
    hipEvent_t start, stop;

    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));
    cublas_vector_scalar_add(handle, d_x, scalar, N);
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));

    printf("------- Vector Scalar Add Kernel ---------\n");
    printf("Elapsed time: %f ms\n", ms);
    printf("------------------------------------------\n");

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    CUBLAS_CHECK(hipblasDestroy(handle));

    return ms;
}