#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>


__global__ void printThreadIndex(int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N)
        printf("BlockIdx.x %d, ThreadIdx.x %d, index %d\n", blockIdx.x, threadIdx.x, index);
}


int main() {

    const int N = 1024;
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    printThreadIndex<<<blocksPerGrid, threadsPerBlock>>>(N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }

    hipDeviceSynchronize();

    return 0;
}
