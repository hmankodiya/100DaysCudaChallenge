// main.cu

#include <iostream>
#include "../include/utils.cuh"
#include "./include/vector_scalar_add.cuh"

int main() {
    // Matrix dimensions
    const int N = 1024;
    const size_t size = N * sizeof(float);

    // Allocate and initialize host matrices
    float val = 100;
    float *h_A = random_normal_clamped_array(N, -1.0f, 1.0f);
    float *h_result;
    h_result = (float *)malloc(size);

    // Allocate device memory
    float *d_A, *d_result;
    CUDA_CHECK(hipMalloc((void **)&d_A, size));
    CUDA_CHECK(hipMalloc((void **)&d_result, size));

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

    // Run the kernel
    float ms = run_vector_scalar_add(d_A, d_result, val, N);

    CUDA_CHECK(hipMemcpy(h_result, d_result, size, hipMemcpyDeviceToHost));
    verify_result(h_result, h_A, val, N);

    // Cleanup
    free(h_A);
    free(h_result);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_result));

    return 0;
}
